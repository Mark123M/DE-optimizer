#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void square_matrix_kernel(const float* matrix, float* result, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width) {
        int idx = row * width + col;
        result[idx] = matrix[idx] * matrix[idx];
    }
}

torch::Tensor square_matrix_cuda(torch::Tensor matrix) {
    const auto height = matrix.size(0);
    const auto width = matrix.size(1);

    auto result = torch::empty_like(matrix);

    dim3 threads_per_block(16, 16);
    dim3 number_of_blocks((width + threads_per_block.x - 1) / threads_per_block.x,
                         (height + threads_per_block.y - 1) / threads_per_block.y);

    square_matrix_kernel<<<number_of_blocks, threads_per_block>>>(
        matrix.data_ptr<float>(), result.data_ptr<float>(), width, height);

    return result;
}

// Defines the operators
TORCH_LIBRARY(diff_evo, m) {
    m.def("square_matrix_cuda(Tensor matrix) -> Tensor");
}

// Registers CUDA implementations for mymuladd, mymul, myadd_out
TORCH_LIBRARY_IMPL(diff_evo, CUDA, m) {
    m.impl("square_matrix_cuda", &square_matrix_cuda);
}